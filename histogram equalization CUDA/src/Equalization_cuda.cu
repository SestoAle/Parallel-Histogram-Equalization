#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : Equalization_cuda.cu
 Author      : 
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "Timer.h"

using namespace cv;
using namespace std;

int width;
int height;
bool with_gpu = true;
string file_name = "dark";

static int eq_CPU(unsigned char *input_ptr);
static int eq_GPU(unsigned char *input_ptr);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)


//Kernel for the color conversion RGB to YCbCr and to compute the histogram of the Y channel.
__global__ void RGB_to_YCbCr_kernel(unsigned char *input, int* hist, int width, int height){

	//Create and initialize a shared histogram to decrease the latency caused by the access to the global memory.
	__shared__ int hist_priv[256];

	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	long point_index;

	//for(int bin_idx = (threadIdx.x*blockDim.x + threadIdx.y); bin_idx < 256; bin_idx += blockDim.x*blockDim.y){
	for(int bin_idx = threadIdx.x; bin_idx < 256; bin_idx += blockDim.x){
		hist_priv[bin_idx] = 0;
	}

	__syncthreads();

	//If doesn't have the required number of threads, the access to the image in global memory is coalesced.
	//The image is saved in a buffer in order to ease the coalesced access;
	for(int i = idx; i < width*height; i += blockDim.x*gridDim.x){
		point_index = i*3;
		int r = input[point_index+0];
		int g = input[point_index+1];
		int b = input[point_index+2];

		int Y = (int) (0.299*r + 0.587*g + 0.114*b);
		int Cb = (int) (128 - 0.168736*r - 0.331264*g +0.5*b);
		int Cr = (int) (128 + 0.5*r - 0.418688*g - 0.081312*b);

		input[point_index+0] = Y;
		input[point_index+1] = Cb;
		input[point_index+2] = Cr;

		//Update the shared histogram.
		atomicAdd(&(hist_priv[Y]), 1);
	}
	__syncthreads();

	//The shared histograms are added to the global histogram.
	for(int bin_idx = threadIdx.x; bin_idx < 256; bin_idx += blockDim.x){
		atomicAdd(&(hist[bin_idx]), hist_priv[bin_idx]);
	}
}

//This kernel equalizes the histogram
__global__ void equalize_kernel(int* cdf, int* hist, int width, int height){

	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	for(int i = idx; i < 256; i += blockDim.x*gridDim.x){
		hist[i] = (int) (((((float)cdf[i] - cdf[0]))/(((float)width*height - 1)))*255);
	}
}

//This kernel maps the new equalized values of the Y channel and
// makes the color conversion from YCbCr to RGB.
__global__ void YCbCr_to_RGB_kernel(unsigned char *input, int* hist, int* cdf, int width, int height){

	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	//long index = (row*width + col);
	long point_index;

	for(int i = idx; i < width*height; i += blockDim.x*gridDim.x){

		point_index = i*3;

		int value_before = input[point_index];
		int value_after = hist[value_before];

		//input[point_index] = value_after;

		int y = value_after;
		int cb = input[point_index+1];
		int cr = input[point_index+2];

		int R = max(0, min(255, (int) (y + 1.402*(cr-128))));
		int G = max(0, min(255, (int) (y - 0.344136*(cb-128) - 0.714136*(cr-128))));
		int B = max(0, min(255, (int) (y + 1.772*(cb- 128))));

		input[point_index+0] = R;
		input[point_index+1] = G;
		input[point_index+2] = B;

	}
}

int main(void)
{
	//Load the image
	cout << "Loading image.." << endl;
	string input_name = "image/" + file_name + ".jpg";
	Mat input = imread(input_name, CV_LOAD_IMAGE_COLOR);

	if(!input.data){
		cout << "Image not found!" << endl;
		return -1;
	}

	height = input.rows;
	width = input.cols;

	//Convert the image into a buffer
	unsigned char *input_ptr = input.ptr();

	cout << "Starting to process.." << endl;

	//Start GPU timer
	GpuTimer timer;
	timer.Start();

	if(with_gpu){
		cout << "Processing with GPU" << endl;
		eq_GPU(input_ptr);
	}else{
		cout << "Processing with CPU" << endl;
		eq_CPU(input_ptr);
	}

	//Stop the GPU timer and show the elapsed time.
	timer.Stop();
	printf("Image equalized in %f msec!\n", timer.Elapsed());

	//Save the image
	cout << "Saving output image.."<< endl;
	string output_name;
	if(with_gpu){
		output_name = "image/" + file_name + "_equalized_gpu.jpg";
	} else{
		output_name = "image/" + file_name + "_equalized_cpu.jpg";
	}
	imwrite(output_name, input);
	cout << "Image saved!"<< endl;

	return 0;
}


//Check the return value of the CUDA runtime API call and exit the application if the call has failed.

static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

static int eq_GPU(unsigned char *input_ptr){

	unsigned char *gpu_input;
	int *d_hist;
	int *d_cdf;
	int *d_hist_eq;
	int h_hist[256] = {0};

	//Allocate the GPU global memory needed.
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpu_input, sizeof(char)*(width*height*3)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_hist, sizeof(int)*(256)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_hist_eq, sizeof(int)*(256)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_cdf, sizeof(int)*(256)));

	//Copy the image buffer to the global memory.
	CUDA_CHECK_RETURN(hipMemcpy(gpu_input, input_ptr, sizeof(char)*(width*height*3), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_hist, h_hist, sizeof(int)*(256), hipMemcpyHostToDevice));

	int block_size = 256;
	int grid_size = (width*height + (block_size-1))/block_size;

	//Call the first kernel.
	RGB_to_YCbCr_kernel<<<grid_size, block_size>>> (gpu_input, d_hist, width, height);

	//Copy to host the histogram computed in the first kernel.
	CUDA_CHECK_RETURN(hipMemcpy(h_hist, d_hist, sizeof(int)*(256), hipMemcpyDeviceToHost));

	int sum = 0;
	int h_cdf[256] = {0};

	for(int i = 0; i < 256; i++){
		sum += h_hist[i];
		h_cdf[i] = sum;
	}

	CUDA_CHECK_RETURN(hipMemcpy(d_cdf, h_cdf, sizeof(int)*(256), hipMemcpyHostToDevice));

	//Call the second kernel.
	equalize_kernel<<<grid_size, block_size>>> (d_cdf, d_hist_eq, width, height);

	//Call the third kernel.
	YCbCr_to_RGB_kernel<<<grid_size, block_size>>> (gpu_input, d_hist_eq, d_cdf, width, height);

	//Copy to host the equalized image.
	CUDA_CHECK_RETURN(hipMemcpy(input_ptr, gpu_input, sizeof(char)*(width*height*3), hipMemcpyDeviceToHost));

	//Release GPU memory.
	CUDA_CHECK_RETURN(hipFree(gpu_input));
	CUDA_CHECK_RETURN(hipFree(d_hist));
	CUDA_CHECK_RETURN(hipFree(d_hist_eq));
	CUDA_CHECK_RETURN(hipFree(d_cdf));

	return 0;
}

//Histogram Equalization with CPU
static int eq_CPU(unsigned char *input_ptr){

	int histogram[256] = {0};

	for (int i = 0; i< height*width*3; i+=3){
		int r = input_ptr[i+0];
		int g = input_ptr[i+1];
		int b = input_ptr[i+2];

		int Y = (int) (0.299*r + 0.587*g + 0.114*b);
		int Cb = (int) (128 - 0.168736*r - 0.331264*g +0.5*b);
		int Cr = (int) (128 + 0.5*r - 0.418688*g - 0.081312*b);

		input_ptr[i+0] = Y;
		input_ptr[i+1] = Cb;
		input_ptr[i+2] = Cr;

		histogram[Y] += 1;
	}

	int sum = 0;
	int histogram_equalized[256] = {0};

	for(int i = 0; i < 256; i++){
		sum += histogram[i];
		histogram_equalized[i] = (int) (((((float)sum - histogram[0]))/(((float)width*height - 1)))*255);

	}

	for (int i = 0; i< height*width*3; i+=3){
		int value_before = input_ptr[i];
		int value_after = histogram_equalized[value_before];

		input_ptr[i] = value_after;

		int y = input_ptr[i+0];
		int cb = input_ptr[i+1];
		int cr = input_ptr[i+2];

		int R = max(0, min(255, (int) (y + 1.402*(cr-128))));
		int G = max(0, min(255, (int) (y - 0.344136*(cb-128) - 0.714136*(cr-128))));
		int B = max(0, min(255, (int) (y + 1.772*(cb- 128))));

		input_ptr[i+0] = R;
		input_ptr[i+1] = G;
		input_ptr[i+2] = B;
	}

	return 0;
}

